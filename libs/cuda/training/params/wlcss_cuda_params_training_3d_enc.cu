#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "../distance.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int32_t *d_mss, *d_mss_offsets, *d_ts, *d_ss, *d_tlen, *d_toffsets, *d_slen, *d_soffsets, *d_params, *d_tmp_windows, *d_tmp_windows_offsets, *d_3d_cost_matrix;
int num_templates, num_streams, num_params_sets, h_ts_length, h_ss_length, h_mss_length, len_h_tmp_windows;

__global__ void wlcss_cuda_kernel(int32_t *d_mss, int32_t *d_mss_offsets, int32_t *d_ts, int32_t *d_ss, int32_t *d_tlen, int32_t *d_toffsets, int32_t *d_slen, int32_t *d_soffsets, int32_t *d_params, int32_t *d_tmp_windows, int32_t *d_tmp_windows_offsets, int32_t *d_3d_cost_matrix){

    int32_t params_idx = threadIdx.x;
    int32_t template_idx = blockIdx.x;
    int32_t stream_idx = blockIdx.y;

    int32_t t_len = d_tlen[template_idx];
    int32_t s_len = d_slen[stream_idx];

    int32_t t_offset = d_toffsets[template_idx];
    int32_t s_offset = d_soffsets[stream_idx];

    int32_t d_mss_offset = d_mss_offsets[params_idx*gridDim.x*gridDim.y+template_idx*gridDim.y+stream_idx];
    int32_t d_tmp_windows_offset = d_tmp_windows_offsets[params_idx*gridDim.x*gridDim.y+template_idx*gridDim.y+stream_idx];

    int32_t *tmp_window = &d_tmp_windows[d_tmp_windows_offset];
    int32_t *mss = &d_mss[d_mss_offset];

    int32_t *t = &d_ts[t_offset];
    int32_t *s = &d_ss[s_offset];

    int32_t reward = d_params[params_idx*3];
    int32_t penalty = d_params[params_idx*3+1];
    int32_t accepteddist = d_params[params_idx*3+2];

    int32_t tmp = 0;

    for(int32_t j=0;j<s_len;j++){
        for(int32_t i=0;i<t_len;i++){
            int32_t distance = d_3d_cost_matrix[s[j]*26 + t[i]];
            if (distance <= accepteddist){
                tmp = tmp_window[i]+reward;
            } else{
                tmp = max(tmp_window[i]-penalty*distance,
                            max(tmp_window[i+1]-penalty*distance,
                                tmp_window[t_len+1]-penalty*distance));
            }
            tmp_window[i] = tmp_window[t_len+1];
            tmp_window[t_len+1] = tmp;
        }
        tmp_window[t_len] = tmp_window[t_len+1];
        mss[j] = tmp_window[t_len+1];
        tmp_window[t_len+1] = 0;
    }
}

extern "C"{
    void wlcss_cuda_init(int32_t *h_tmp_windows_offsets,
                         int32_t *h_mss_offsets, 
                         int32_t *h_ts, int32_t *h_ss, 
                         int32_t *h_tlen, int32_t *h_toffsets, 
                         int32_t *h_slen, int32_t *h_soffsets, 
                         int num_ts, int num_ss, 
                         int num_ps, int h_ts_len, int h_ss_len, int h_mss_len){

        num_templates = num_ts;
        num_streams = num_ss;
        num_params_sets = num_ps;
        h_ts_length = h_ts_len;
        h_ss_length = h_ss_len;
        h_mss_length = h_mss_len;


        //Allocate memory for cost matrix
        gpuErrchk( hipMalloc((void **) &d_3d_cost_matrix, 676 * sizeof(int32_t)) );
        gpuErrchk( hipMemcpy(d_3d_cost_matrix, h_3d_cost_matrix, 676 * sizeof(int32_t), hipMemcpyHostToDevice) );
        
        // Allocate memory for templates array
        gpuErrchk( hipMalloc((void **) &d_ts, h_ts_length * sizeof(int32_t)) );
        gpuErrchk( hipMemcpy(d_ts, h_ts, h_ts_length * sizeof(int32_t), hipMemcpyHostToDevice) );

        //Allocate memory for templates lengths
        gpuErrchk( hipMalloc((void **) &d_tlen, num_templates * sizeof(int32_t)) );
        gpuErrchk( hipMemcpy(d_tlen, h_tlen, num_templates * sizeof(int32_t), hipMemcpyHostToDevice) );

        // Allocate memory for templates offsets
        gpuErrchk( hipMalloc((void **) &d_toffsets, num_templates * sizeof(int32_t)) );
        gpuErrchk( hipMemcpy(d_toffsets, h_toffsets, num_templates * sizeof(int32_t), hipMemcpyHostToDevice) );

        // Allocate memory for streams array
        gpuErrchk( hipMalloc((void **) &d_ss, h_ss_length * sizeof(int32_t)) );
        gpuErrchk( hipMemcpy(d_ss, h_ss, h_ss_length * sizeof(int32_t), hipMemcpyHostToDevice) );

        // Allocate memory for streams lengths
        gpuErrchk( hipMalloc((void **) &d_slen, num_streams * sizeof(int32_t)) );
        gpuErrchk( hipMemcpy(d_slen, h_slen, num_streams * sizeof(int32_t), hipMemcpyHostToDevice) );

        // Allocate memory for streams offsets
        gpuErrchk( hipMalloc((void **) &d_soffsets, num_streams * sizeof(int32_t)) );
        gpuErrchk( hipMemcpy(d_soffsets, h_soffsets, num_streams * sizeof(int32_t), hipMemcpyHostToDevice) );

        // Allocate memory for matching scores
        gpuErrchk( hipMalloc((void **) &d_mss, h_mss_length * sizeof(int32_t)) );

        //Allocate memory for matching scores offsets
        gpuErrchk( hipMalloc((void **) &d_mss_offsets, num_streams*num_templates*num_params_sets * sizeof(int32_t)) );
        gpuErrchk( hipMemcpy(d_mss_offsets, h_mss_offsets, num_streams*num_templates*num_params_sets * sizeof(int32_t), hipMemcpyHostToDevice) );

        // Allocate memory for d_params
        gpuErrchk( hipMalloc((void **) &d_params, num_params_sets * 3 * sizeof(int32_t)) );
        
        // Allocate memory for tmp_windows
        len_h_tmp_windows = (h_ts_len + 2 * num_templates) * num_params_sets * num_streams;
        gpuErrchk( hipMalloc((void **) &d_tmp_windows, len_h_tmp_windows * sizeof(int32_t)) );
        
        int len_h_tmp_windows_offsets = num_templates * num_params_sets * num_streams;
        gpuErrchk( hipMalloc((void **) &d_tmp_windows_offsets, len_h_tmp_windows_offsets * sizeof(int32_t)) );
        gpuErrchk( hipMemcpy(d_tmp_windows_offsets, h_tmp_windows_offsets, len_h_tmp_windows_offsets * sizeof(int32_t), hipMemcpyHostToDevice) );

    }

    void wlcss_cuda(int32_t *h_params, int32_t *h_mss, int32_t *h_tmp_windows){

        gpuErrchk( hipMemcpy(d_params, h_params, num_params_sets * 3 * sizeof(int32_t), hipMemcpyHostToDevice) );
        gpuErrchk( hipMemcpy(d_mss, h_mss, h_mss_length * sizeof(int32_t), hipMemcpyHostToDevice) );
        gpuErrchk( hipMemcpy(d_tmp_windows, h_tmp_windows, len_h_tmp_windows * sizeof(int32_t), hipMemcpyHostToDevice) );

        wlcss_cuda_kernel<<<dim3(num_templates, num_streams), num_params_sets>>>(d_mss, d_mss_offsets, d_ts, d_ss, d_tlen, d_toffsets, d_slen, d_soffsets, d_params, d_tmp_windows, d_tmp_windows_offsets, d_3d_cost_matrix);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        gpuErrchk( hipMemcpy(h_mss, d_mss, h_mss_length * sizeof(int32_t), hipMemcpyDeviceToHost) );
    }
    
    void wlcss_freemem(){
        
        hipFree(d_ts);
        hipFree(d_tlen);
        hipFree(d_toffsets);
        
        hipFree(d_ss);
        hipFree(d_slen);
        hipFree(d_soffsets);
        
        hipFree(d_mss);
        hipFree(d_mss_offsets);
        hipFree(d_params);
        
        hipFree(d_tmp_windows);
        hipFree(d_tmp_windows_offsets);

        hipFree(d_3d_cost_matrix);
    }
}
